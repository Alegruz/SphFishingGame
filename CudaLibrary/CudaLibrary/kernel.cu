#include "hip/hip_runtime.h"
﻿#include <cstdlib>
#include <cstdio>
#include <string.h>

#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_impl.cuh"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


//particleSystem_cuda.cu
extern "C"
{

    void cudaInit(int argc, char** argv)
    {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        devID = findCudaDevice(argc, (const char**)argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void allocateArray(void** devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void* devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void threadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    void copyArrayToDevice(void* device, const void* host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char*)device + offset, host, size, hipMemcpyHostToDevice));
    }

    void setParameters(SimParams* hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }


    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint& numBlocks, uint& numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }



    void integrateSystem(float* pos,
        float* vel,
        float deltaTime,
        uint numParticles)
    {
        thrust::device_ptr<float4> d_pos4((float4*)pos);
        thrust::device_ptr<float4> d_vel4((float4*)vel);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4 + numParticles, d_vel4 + numParticles)),
            integrate_functor(deltaTime));
    }

    void calcHash(uint* gridParticleHash,
        uint* gridParticleIndex,
        float* pos,
        int    numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD << < numBlocks, numThreads >> > (gridParticleHash,
            gridParticleIndex,
            (float4*)pos,
            numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void reorderDataAndFindCellStart(uint* cellStart,
        uint* cellEnd,
        float* sortedPos,
        float* sortedVel,
        uint* gridParticleHash,
        uint* gridParticleIndex,
        float* oldPos,
        float* oldVel,
        uint   numParticles,
        uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells * sizeof(uint)));

        uint smemSize = sizeof(uint) * (numThreads + 1);
        reorderDataAndFindCellStartD << < numBlocks, numThreads, smemSize >> > (
            cellStart,
            cellEnd,
            (float4*)sortedPos,
            (float4*)sortedVel,
            gridParticleHash,
            gridParticleIndex,
            (float4*)oldPos,
            (float4*)oldVel,
            numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

    }

    void computeDensityAndPressure(float* newDensity, float* newPressure,
        float* sortedPos,
        uint* gridParticleIndex,
        uint* cellStart,
        uint* cellEnd,
        uint   numParticles,
        uint   numCells)
    {

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        computeDensityAndPressureDevice << < numBlocks, numThreads >> > (newDensity, newPressure,
            (float4*)sortedPos,
            gridParticleIndex,
            cellStart,
            cellEnd,
            numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

    }

    void computeForces(float* newVelocities,
        float* newForce,
        float deltaTime,
        float* oldPos,               // input: sorted positions
        float* oldVel,
        float* densities,
        float* pressures,
        uint* gridParticleIndex,    // input: sorted particle indices
        uint* cellStart,
        uint* cellEnd,
        uint    numParticles,
        uint   numCells)
    {
        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        computeForceDevice <<< numBlocks, numThreads >>> ((float4*)newVelocities,
            (float4*)newForce,
            deltaTime,
            (float4*)oldPos,               // input: sorted positions
            (float4*)oldVel,
            densities,
            pressures,
            gridParticleIndex,    // input: sorted particle indices
            cellStart,
            cellEnd,
            numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }


    void sortParticles(uint* dGridParticleHash, uint* dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
            thrust::device_ptr<uint>(dGridParticleIndex));
    }

}   // extern "C"
